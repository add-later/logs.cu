#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void scaleVector(const float *input, float *output, float scale, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        output[idx] = input[idx] * scale;
    }
}

int main() {
    int N = 1024;
    size_t size = N * sizeof(float);

    // host memory
    float *h_input = (float*)malloc(size);
    float *h_output = (float*)malloc(size);

    for (int i=0; i<N; i++) {
        h_input[i] = i * 1.0f;
    }

    // device (global) memory
    float *d_input, *d_output;
    hipMalloc(&d_input, size);
    hipMalloc(&d_output, size);

    hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);

    int threadsPerBlock=256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    scaleVector<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, 2.0f, N);
    hipDeviceSynchronize();

    hipMemcpy(h_output, d_output, size, hipMemcpyDeviceToHost);

    for(int i=0; i<10;i++) {
        printf("h_output[%d] = %f\n", i, h_output[i]);
    }

    hipFree(d_input);
    hipFree(d_output);

    free(h_input);
    free(h_output);
    return 0;
}