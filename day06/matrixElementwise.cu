#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void matrixAddElem(float *A, float *B, float *C, int N, int M){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < M) {
        int index = row * M + col;
        C[index] = A[index] + B[index];
    }
}

int main() {
   int M = 2;
    int N = 2;

    int size = M * N * sizeof(float);
    
    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);

    for (int i = 1; i <= 4; i++) {
        h_A[i-1] = i;
    }
    for (int i = 5; i <= 8; i++) {
        h_B[i-5] = i;
    }
    
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(8, 8);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                        (M + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matrixAddElem<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N, M);
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    printf("Matrix A:\n");
    printf("%.2f %.2f\n%.2f %.2f\n\n", h_A[0], h_A[1], h_A[2], h_A[3]);
    
    printf("Matrix B:\n");
    printf("%.2f %.2f\n%.2f %.2f\n\n", h_B[0], h_B[1], h_B[2], h_B[3]);
    
    printf("Matrix C (result):\n");
    printf("%.2f %.2f\n%.2f %.2f\n\n", h_C[0], h_C[1], h_C[2], h_C[3]);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
    
    return 0;
}